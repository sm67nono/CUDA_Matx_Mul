#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "testMultiGPU_Jacobi2D_Decom.cuh"
#include <iostream>
#include <chrono>
#include <memory>
#include <vector>
using namespace std;
using namespace std::chrono;

#define IMUL(a,b) __mul24(a,b)


//hipError_t performMultiGPUJacobi();

//Support for below c++14 on *nix
template<typename T, typename ...Args>
std::unique_ptr<T> make_unique(Args&& ...args)
{
	return std::unique_ptr<T>(new T(std::forward<Args>(args)...));
}

struct create_Device
{
	int deviceID;

	//In a GPU topology set the GPU position
	int devicePosition_X;
	int devicePosition_Y;
	int devicePosition_Z;

	vector<float> eHalo;
	vector<float> wHalo;
	vector<float> nHalo;
	vector<float> sHalo;

};



//Simple Jacobi iteration
__global__ void jacobi_Simple(const float *A0, const float *A1, const float *A2, const float *A3, const float *A4, float *x_in, float *x_out, const float *rhs, float *nhalo, float *shalo, const int deviceID, const int numDevices)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	float result = rhs[index];

	int dim_x = blockDim.x;// dim across x
	int dim_y = gridDim.x;
	int x_pos = blockIdx.x;
	int y_pos = threadIdx.x;


	//result = nhalo[y_pos];
	//x_out[index] = result;
	//Get the boundaries

	int leftBoundaryElem = x_pos * (dim_x);

	int rightBoundaryElem = (x_pos * dim_x) + (dim_x - 1);

	int topBoundaryElem = y_pos + ((dim_y - 1) * (dim_x));

	int bottomBoundaryElem = y_pos;


	//Halo computation for 1D Decompostion: For the First and Last GPU Halo computation on both the sides(nhalo and shalo wont be needed)
	if (numDevices > 1)
	{
		//First GPU
		if (deviceID == 0) {
			//We need to use nhalos

			//Carry out computations for boundary elements
			if (index != leftBoundaryElem)
				//Left
				result -= A1[index] * x_in[index - 1];

			if (index != rightBoundaryElem)
				//Right 
				result -= A3[index] * x_in[index + 1];
			if (index != bottomBoundaryElem)
				//Bottom
				result -= A0[index] * x_in[index - dim_x];

			if (index != topBoundaryElem)
				//Top
				result -= A4[index] * x_in[index + dim_x];
			//The top boundary needs element from nhalo
			if (index == topBoundaryElem)
				//nHalos
				result -= A4[index] * nhalo[y_pos];




			result /= A2[index];

			x_out[index] = result;


			//Update Halo at the end of computation
			if (index == topBoundaryElem)
				//nHalos updated
				nhalo[y_pos] = result;

			return;

		}

		//Last GPU
		else if (deviceID == (numDevices - 1)) {
			//We need to use shalos

			//Carry out computations for boundary elements
			if (index != leftBoundaryElem)
				//Left
				result -= A1[index] * x_in[index - 1];

			if (index != rightBoundaryElem)
				//Right 
				result -= A3[index] * x_in[index + 1];
			if (index != bottomBoundaryElem)
				//Bottom
				result -= A0[index] * x_in[index - dim_x];
			//The Bottom boundary needs elements from shalo
			if (index == bottomBoundaryElem)
				//nHalos
				result -= A0[index] * shalo[y_pos];


			if (index != topBoundaryElem)
				//Top
				result -= A4[index] * x_in[index + dim_x];


			result /= A2[index];

			x_out[index] = result;

			//Update Halo at the end of computation
			if (index == bottomBoundaryElem)
				//sHalos updated
				shalo[y_pos] = result;

			return;

		}
		//For all the middle GPUs
		else
		{
			//We need to use both shalos and nhalos

			//Carry out computations for boundary elements
			if (index != leftBoundaryElem)
				//Left
				result -= A1[index] * x_in[index - 1];

			if (index != rightBoundaryElem)
				//Right 
				result -= A3[index] * x_in[index + 1];

			if (index != bottomBoundaryElem)
				//Bottom
				result -= A0[index] * x_in[index - dim_x];
			//The Bottom boundary needs elements from shalo
			if (index == bottomBoundaryElem)
				//nHalos
				result -= A0[index] * shalo[y_pos];


			if (index != topBoundaryElem)
				//Top
				result -= A4[index] * x_in[index + dim_x];
			//The top boundary needs element from nhalo
			if (index == topBoundaryElem)
				//nHalos
				result -= A4[index] * nhalo[y_pos];





			result /= A2[index];

			x_out[index] = result;



			//Update Halo at the end of computation
			if (index == bottomBoundaryElem)
				//sHalos updated
				shalo[y_pos] = result;

			//Update Halo at the end of computation
			if (index == topBoundaryElem)
				//nHalos updated
				nhalo[y_pos] = result;



			return;

		}

	}

	//For computations on a Machine with a single GPU
	else
	{
		{//For some reason order of computation (left,right,top and bottom) gives a different result

		 //Carry out computations for boundary elements
			if (index != leftBoundaryElem)
				//Left
				result -= A1[index] * x_in[index - 1];

			if (index != rightBoundaryElem)
				//Right 
				result -= A3[index] * x_in[index + 1];
			if (index != bottomBoundaryElem)
				//Bottom
				result -= A0[index] * x_in[index - dim_x];

			if (index != topBoundaryElem)
				//Top
				result -= A4[index] * x_in[index + dim_x];



			result /= A2[index];

			x_out[index] = result;

			return;
		}
	}



}




//====================================Creating Topology with the number of Devices available====================================

void generateGPUGRID(unsigned int numDevices, int &numberOfDevicesAlong_X, int &numberOfDevicesAlong_Y)
{
	//Finding GPU topology along x and y
	//Assumuing total number of devices is a perfect square(To be changed later)
	numberOfDevicesAlong_X = (int)sqrt(numDevices);
	numberOfDevicesAlong_Y =(int) numberOfDevicesAlong_X;
}


/* Creates a topology for a number of devices in a system
for ex. The devices are aware of left, right, top and bottom neigbours in 2D
1. It also decides the chunk per devices by determining x-dimension and y-dimensions for per chunk of data per device.
2. It also initializes halos for each devices which can be exchanged with the neighbours
*/

void createTopology(unsigned numDevices, vector<create_Device> &deviceArray, int numberOfDevicesAlong_X, int numberOfDevicesAlong_Y)
{

	deviceArray.resize(numDevices);
	unsigned int deviceCount = 0;
	for (int gridCount_X = 0; gridCount_X < numberOfDevicesAlong_X; gridCount_X++) {
		for (int gridCount_Y = 0; gridCount_Y < numberOfDevicesAlong_Y; gridCount_Y++) {
			deviceArray[deviceCount].deviceID = deviceCount;
			deviceArray[deviceCount].devicePosition_X = gridCount_X;
			deviceArray[deviceCount].devicePosition_Y = gridCount_Y;
			//devicePosition_Z to be changed later
			deviceArray[deviceCount].devicePosition_Z = 1;
			deviceCount++;
		}
	}


}
//==============================================================================================================================

//Init Halos: In 1D decomposition only North and South Halos are used. In 2D decomposition North, South, East and West Halo need to be initialized and computed
//In 3D decomposition North, South, East , West, Top and Bottom needs to be initialized and computed
void initHalos(int numDevices, vector<create_Device> &deviceArray, int dim_x, float *vec_in) {


	deviceArray.resize(numDevices);
	int chunksize = ((dim_x*dim_x) / numDevices);
	cout << "chunk size is :" << chunksize << endl;
	for (int i = 0, pos = chunksize; i < numDevices; pos += chunksize, i++) {

		deviceArray[i].deviceID = i;
		deviceArray[i].nHalo.resize(dim_x);
		//TODO: 2D halo exchange
		//TODO: deviceArray[i].eHalo.resize(dim_x);
		//TODO: deviceArray[i].wHalo.resize(dim_x);
		deviceArray[i].sHalo.resize(dim_x);

		if (numDevices == 1)
		{
			for (int count = 0; count < dim_x; count++)
			{

				deviceArray[i].nHalo[count] = 1.0f;
				deviceArray[i].sHalo[count] = 1.0f;
			}
			return;
		}

		//First Device needs only nHalo
		if (i == 0)
		{

			for (int k = pos, count = 0; count < dim_x; k++, count++)
			{
				cout << "Halo nPosition for first Device is : " << k << endl;
				deviceArray[i].nHalo[count] = vec_in[k];
			}

		}

		//Last device needs only sHalo
		else if (i == (numDevices - 1))
		{

			for (int k = pos - (chunksize + dim_x), count = 0; count < dim_x; count++, k++)
			{
				cout << "Halo sPosition for Last Device is : " << k << endl;
				deviceArray[i].sHalo[count] = vec_in[k];
			}

		}

		//All the other devices need both sHalo and nHalo
		else
		{


			for (int k = pos, count = 0; count < dim_x; count++, k++)
			{
				cout << "Halo nPosition for Mid Device " << i << " is : " << k << endl;
				deviceArray[i].nHalo[count] = vec_in[k];
			}
			for (int k = pos - (chunksize + dim_x), count = 0; count < dim_x; count++, k++)
			{
				cout << "Halo sPosition for Mid Device " << i << "  is : " << k << endl;
				deviceArray[i].sHalo[count] = vec_in[k];
			}


		}

	}


}

//TODO:Create a Halo Exchange Mechanism for 2D Multi GPU topology
void initHalos2D(create_Device &device, int chunk_X, int chunk_Y, float *vec_in, int maxdevicesAlong_X, int maxDevicesAlong_Y, int rowStartPos, int rowEndPos, int dim) {

	cout << endl <<"Inside Halo Computation 2D. printing Details" ;
	cout << endl <<"Device ID "<< device.deviceID;
	cout << endl <<"Device position X " << device.devicePosition_X;
	cout << endl <<"Device position Y " << device.devicePosition_Y;
	cout << endl << "Row Start " << rowStartPos;
	cout << endl << "Row End " << rowEndPos;

	//Checks provided for Boundary devices in GPU topology
	if ((device.devicePosition_Y - 1) >= 0) {
		cout << "West Halo needed ";
		device.wHalo.resize(chunk_Y);
		for (int rowNum = 0; rowNum < chunk_Y; rowNum++)
		{
			device.wHalo[rowNum] = vec_in[rowStartPos];
			//cout << rowStartPos << " ";
			rowStartPos += dim;	
		}

	}

	if ((device.devicePosition_Y + 1) < maxdevicesAlong_X) {
		cout << "East Halo needed  ";
		device.eHalo.resize(chunk_Y);
		for (int rowNum = 0; rowNum < chunk_Y; rowNum++)
		{
			device.eHalo[rowNum] = vec_in[rowEndPos];
			//cout << rowEndPos << " ";
			rowEndPos += dim;
		}
	}
	if ((device.devicePosition_X - 1) >= 0) {
		cout << "South Halo needed ";
		device.sHalo.resize(chunk_X);
		for (int rowNum = 0; rowNum < chunk_X; rowNum++)
		{
			device.sHalo[rowNum] = vec_in[rowStartPos];
			cout << rowStartPos << " ";
			rowStartPos ++;
		}
		
	}
	if ((device.devicePosition_X + 1) < maxDevicesAlong_Y) {
		cout << "North Halo needed  ";
		device.sHalo.resize(chunk_X);
		rowStartPos = rowStartPos + (dim * chunk_Y);
		for (int rowNum = 0; rowNum < chunk_X; rowNum++)
		{
			device.sHalo[rowNum] = vec_in[rowStartPos];
			cout << rowStartPos << " ";
			rowStartPos++;
		}
	}


}



//Domain Decompostion 2D:Data distribution


//Init matrix Diagonals A0, A1, A2, A3, A4
void copyValues(float *A0, float *A1, float *A2, float *A3, float *A4, float *rhs, float *vec_in, float *vec_out, int dim, float *val_A0, float *val_A1, float *val_A2, float *val_A3, float *val_A4, float *val_rhs, float *val_x_in)
{

	unsigned int size = dim * dim;

	for (unsigned int i = 0; i < size; i++)
	{
		A0[i] = i;// val_A0[i];
		A1[i] = val_A1[i];
		A2[i] = val_A2[i];
		A3[i] = val_A3[i];
		A4[i] = val_A4[i];
		rhs[i] = val_rhs[i];
		vec_in[i] = val_x_in[i];
		vec_out[i] = 0.0f;

	}



}


void getAllDeviceProperties() {

	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		cout << " Device Number: " << i << endl;
		cout << " Device name: " << prop.name << endl;
		cout << " Memory Clock Rate (KHz): " << prop.memoryClockRate << endl;
		cout << " Memory Bus Width (bits): " << prop.memoryBusWidth << endl;;
		cout << " Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6 << endl << endl << endl;
	}
}



hipError_t performMultiGPUJacobi(unsigned int val_dim, unsigned int numJacobiIt, float* val_A0, float* val_A1, float* val_A2, float* val_A3, float* val_A4, float* val_rhs, float* val_x_in)
{
	//Fixed value changed later
	int dim = 8;
	if (val_dim != 0) {
		dim = val_dim;
	}


	//TODO: write a 2D domain decomposition method for more than 2 GPUs
	int size = dim * dim;

	//auto result = make_unique<float[]>(size);

	//Create Diagonal Vectors
	std::vector<float> a0(size);
	std::vector<float> a1(size);
	std::vector<float> a2(size);
	std::vector<float> a3(size);
	std::vector<float> a4(size);
	std::vector<float> vec_in(size);
	std::vector<float> vec_out(size);
	std::vector<float> rhs(size);
	std::vector<float> result(size);

	//Used for exchanging the Halos after each Jacobi Iteration
	std::vector<float> prev_nHalo(dim);
	std::vector<float> curr_sHalo(dim);

	//Get the total number of devices
	int numDevices;
	hipGetDeviceCount(&numDevices);
	cout << endl << "Total number of Devices in the System are :  " << numDevices << endl;

	getAllDeviceProperties();


	//Configuring the number of GPU's manually
	//numDevices=2;

	copyValues(&a0[0], &a1[0], &a2[0], &a3[0], &a4[0], &rhs[0], &vec_in[0], &vec_out[0], dim, &val_A0[0], &val_A1[0], &val_A2[0], &val_A3[0], &val_A4[0], &val_rhs[0], &val_x_in[0]);

	vector<create_Device> deviceArray;



	/* Distributed Compuation using Halos: Algorithm

	1. Init Halos.
	1.a) In 1D decomposition nhalo and shalo intialized from vector x_in
	1.b) In 2D decompsition nhalo,shalo, ehalo and whalo initialozed from vector x_in
	2. Pass the halos to Jacobi_kernal.
	3. Store the result computed at the boundary into the halo boundary positions.
	4. Swap nhalo and shalo pairs in 1D decompostion. Swap (nhalo,shalo) and (ehalo,whalo) in 2D.

	*/

	/*initHalos(numDevices, deviceArray, dim, &vec_in[0]);

	//Display Halos
	if (numDevices > 1) {
		cout << endl << "Halo Init.." << endl;

		for (int i = 0; i < numDevices; i++) {

			cout << "Device ID: " << deviceArray[i].deviceID;

			//First Device needs only nHalo
			if (i == 0)
			{
				cout << "First Device";
				for (int k = 0; k < dim; k++)
				{
					cout << deviceArray[i].nHalo[k];
				}

			}

			//Last device needs only sHalo
			else if (i == (numDevices - 1))
			{
				cout << "Last Device";
				for (int k = 0; k < dim; k++)
				{
					cout << deviceArray[i].sHalo[k];
				}

			}

			//All the other devices need both sHalo and nHalo
			else
			{

				cout << "Middle Device";
				for (int k = 0; k < dim; k++)
				{
					cout << deviceArray[i].nHalo[k];
				}

				for (int k = 0; k < dim; k++)
				{
					cout << deviceArray[i].sHalo[k];
				}


			}
			cout << endl;


		}

		cout << endl;
		cout << endl;
		cout << endl;

	}

	cout << "A0             ....";
	for (int i = 0; i < size; i++) {
		cout << a0[i] << " ";
	}
	cout << endl;

	cout << "A1             ....";
	for (int i = 0; i < size; i++) {
		cout << a1[i] << " ";
	}
	cout << endl;
	cout << "A2             ....";
	for (int i = 0; i < size; i++) {
		cout << a2[i] << " ";
	}
	cout << endl;
	cout << "A3             ....";
	for (int i = 0; i < size; i++) {
		cout << a3[i] << " ";
	}
	cout << endl;
	cout << "A4             ....";
	for (int i = 0; i < size; i++) {
		cout << a4[i] << " ";
	}
	cout << endl;

	cout << "RHS             ....";
	for (int i = 0; i < size; i++) {
		cout << rhs[i] << " ";
	}
	cout << endl;

	cout << "Vec In            ...." << endl;

	for (int i = size - 1; i >= 0; i--) {


		if ((i + 1) % dim == 0) { cout << endl; }

		cout << vec_in[i] << " ";
	}

	cout << endl;



	cout << "Made it here.."; */

	//=================================Domain Decomposition Logic Starts=================================================================

	/*Generating a GPU Grid with  multiple GPUs and creating a Topology*/

	int numberOfDevicesAlong_X = 1;
	int numberOfDevicesAlong_Y = 1;
	generateGPUGRID(numDevices, numberOfDevicesAlong_X, numberOfDevicesAlong_Y);
	cout << "GPU grid structure is : " << numberOfDevicesAlong_X << " X " << numberOfDevicesAlong_Y << endl;

	/* Creating a GPU topology with multiple devices*/
	createTopology(numDevices, deviceArray, numberOfDevicesAlong_X, numberOfDevicesAlong_Y);


	//Set Decomposition dimension 1D or 2D
	int decom_Dim = 2;

	//Allocate memory on the devices

	//Let the total number of GPU be 2 : has to be changed later
	//Computation divided into (size/2) on first and size-(size/2) on second
	int *domainDivision;
	domainDivision = new int[numDevices];



	//Logic for total chunk per device (Domain distribution)
	for (int i = 0; i < numDevices; i++) {
		//Chunk per GPU will be same irrepective of 1D or 2D decomposition
		domainDivision[i] = size / numDevices;
	}


	//For use on Device 
	float *d_A0[4],
		*d_A1[4],
		*d_A2[4],
		*d_A3[4],
		*d_A4[4],
		*d_Vec_In[4],
		*d_Vec_Out[4],
		*d_Rhs[4],
		*d_nhalos[4],
		*d_shalos[4],
		*d_ehalos[4],
		*d_whalos[4];

	/* The domain division is done in 1D rowise */
	for (int dev = 0; dev < numDevices; dev++)
	{
		//Setting the device before allocation
		hipSetDevice(dev);

		//cudamalloc the Diagonals
		hipMalloc((void**)&d_A0[dev], domainDivision[dev] * sizeof(float));
		hipMalloc((void**)&d_A1[dev], domainDivision[dev] * sizeof(float));
		hipMalloc((void**)&d_A2[dev], domainDivision[dev] * sizeof(float));
		hipMalloc((void**)&d_A3[dev], domainDivision[dev] * sizeof(float));
		hipMalloc((void**)&d_A4[dev], domainDivision[dev] * sizeof(float));

		//cudamalloc the Input Vector and Result vector
		hipMalloc((void**)&d_Vec_In[dev], domainDivision[dev] * sizeof(float));
		hipMalloc((void**)&d_Vec_Out[dev], domainDivision[dev] * sizeof(float));
		hipMalloc((void**)&d_Rhs[dev], domainDivision[dev] * sizeof(float));

		//hipMalloc Halos: North and South--1D. TODO: East and West for 2D
		hipMalloc((void**)&d_nhalos[dev], (dim / decom_Dim) * sizeof(float));
		hipMalloc((void**)&d_shalos[dev], (dim / decom_Dim) * sizeof(float));
		hipMalloc((void**)&d_ehalos[dev], (dim / decom_Dim) * sizeof(float));
		hipMalloc((void**)&d_whalos[dev], (dim / decom_Dim) * sizeof(float));

	}




	/* The transfer of Data from Host to Device :  Domain Decomposition in 1D*/
	if (decom_Dim == 1) {

		for (int dev = 0, pos = 0; dev < numDevices; pos += domainDivision[dev], dev++)
		{
			//Setting the device before allocation
			hipSetDevice(dev);

			//Copy the diagonals from host to device
			hipMemcpy(d_A0[dev], &a0[0] + pos, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_A1[dev], &a1[0] + pos, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_A2[dev], &a2[0] + pos, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_A3[dev], &a3[0] + pos, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_A4[dev], &a4[0] + pos, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);

			//Copy in and out vectors and RHS
			hipMemcpy(d_Vec_In[dev], &vec_in[0] + pos, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vec_Out[dev], &vec_out[0] + pos, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Rhs[dev], &rhs[0] + pos, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);

			//Copy intial Halos in 1D : TODO compute more than 1D
			if (dev == 0) {
				hipMemcpy(d_nhalos[dev], &deviceArray[dev].nHalo[0], dim * sizeof(float), hipMemcpyHostToDevice);
			}
			else if (dev == (numDevices - 1)) {
				hipMemcpy(d_shalos[dev], &deviceArray[dev].sHalo[0], dim * sizeof(float), hipMemcpyHostToDevice);
			}
			else {
				hipMemcpy(d_nhalos[dev], &deviceArray[dev].nHalo[0], dim * sizeof(float), hipMemcpyHostToDevice);
				hipMemcpy(d_shalos[dev], &deviceArray[dev].sHalo[0], dim * sizeof(float), hipMemcpyHostToDevice);
			}
		}
	}



	/* The transfer of Data from Host to Device :  Domain Decomposition in 2D*/
	if (decom_Dim == 2) {
		
		//Total elements along each dim in 2D
		int chunk_X = dim / numberOfDevicesAlong_X;
		int chunk_Y = dim / numberOfDevicesAlong_Y;


		//Create Partial Diagonal Vectors
		//Size per GPU will be
		int chunkSize = chunk_X * chunk_Y;
		std::vector<float> partial_a0(chunkSize);
		std::vector<float> partial_a1(chunkSize);
		std::vector<float> partial_a2(chunkSize);
		std::vector<float> partial_a3(chunkSize);
		std::vector<float> partial_a4(chunkSize);
		std::vector<float> partial_vec_in(chunkSize);
		std::vector<float> partial_vec_out(chunkSize);
		std::vector<float> partial_rhs(chunkSize);
		std::vector<float> partial_result(chunkSize);


		for (int dev = 0; dev < numDevices; dev++)
		{

			//Test the properties of the device assigned
			cout << endl << "New Logical Device created " << deviceArray[dev].deviceID;
			cout << endl << "New Logical Device (X,Y) coord (" << deviceArray[dev].devicePosition_X << "," << deviceArray[dev].devicePosition_Y << ")";


			//==========Important: Logic for creation of Chunks to be allocated to GPUs==========================================

			//Important : Mention about the correlation between the topology and data position in the thesis
			hipSetDevice(dev);
			int devicePosX = deviceArray[dev].devicePosition_X;
			int devicePosY = deviceArray[dev].devicePosition_Y;

			//Calculating data position based on device coords
			//numberOfDevicesAlong_X * Chunk_X * Chunk_Y : finds out the  total data per row of GPUs allocated
			int dataStartPos_X = (devicePosX * numberOfDevicesAlong_X * chunk_X * chunk_Y) + (devicePosY * chunk_X);
			int dataEndPos_X = dataStartPos_X + chunk_X;

			//One complete row across all GPU is dim in order to get the next element above an element we add (currentPosition + dim )
			int rowStartPos = dataStartPos_X;
			int rowEndPos = dataEndPos_X;
			int indexCounter = 0;
			//Initialize Halos
			initHalos2D(deviceArray[dev], chunk_X, chunk_Y, &vec_in[0], numberOfDevicesAlong_X, numberOfDevicesAlong_Y, rowStartPos, rowEndPos-1, dim);
			for (int rowNum = 0; rowNum < chunk_Y; rowNum++)
			{
				//Get one complete row for the GPU
				for (int pos = rowStartPos; pos < rowEndPos; pos++)
				{
					partial_a0[indexCounter] = a0[pos];
					partial_a1[indexCounter] = a1[pos];
					partial_a2[indexCounter] = a2[pos];
					partial_a3[indexCounter] = a3[pos];
					partial_a4[indexCounter] = a4[pos];
					partial_vec_in[indexCounter] = vec_in[pos];
					partial_vec_out[indexCounter] = vec_out[pos];
					partial_rhs[indexCounter] = rhs[pos];
					partial_result[indexCounter] = result[pos];
					indexCounter++;
				}
				rowStartPos += dim;
				rowEndPos += dim;
			}

		

			//==========Important: Logic for creation of Chunks to be allocated to GPUs Ends ==========================================
			
			//Testing if inputs are correct
			 /*cout << endl << endl;
			for (int i = 0; i < indexCounter; i++) {
				if ((i%chunk_X) == 0)cout << endl;
				cout << partial_a0[i]<<" ";
			}*/


			//Copy the diagonals from host to device : calling all at once instead of putting inside the for loop
			hipMemcpy(d_A0[dev], &partial_a0, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_A1[dev], &partial_a1, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_A2[dev], &partial_a2, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_A3[dev], &partial_a3, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_A4[dev], &partial_a4, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);

			//Copy in and out vectors and RHS
			hipMemcpy(d_Vec_In[dev], &partial_vec_in, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vec_Out[dev], &partial_vec_out, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Rhs[dev], &partial_rhs, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);

			//Copy intial Halos in 2D
			
		}
	}
	//=================================Domain Decomposition Logic Ends =================================================================


	if (auto err = hipGetLastError())
	{
		cout << "Jacobi launch failed: " << hipGetErrorString(err) << endl;
		return err;
	}



	if (auto err = hipGetLastError())
	{
		cout << "Jacobi launch failed: " << hipGetErrorString(err) << endl;
		return err;
	}




	if (auto err = hipGetLastError())
	{
		cout << "Jacobi launch failed: " << hipGetErrorString(err) << endl;
		return err;
	}

	//multMatrix(d_A0, d_A1, d_A2, d_A3, d_A4, myDim, d_vec, d_res);

	//Perform one Jacobi Step
	int blocksize = dim / numDevices; //TODO: make it to more than 2 GPUs
	int threads = dim;

	//Call to kernal
	int iterations = 4;
	if (numJacobiIt != 0) {
		iterations = numJacobiIt;
	}

	for (int i = 0; i < iterations; i++)
	{

		cout << endl << endl << "Iteration : " << i + 1 << endl << endl << endl;

		//TODO: optimization using kernel instead of For Loop
		for (int dev = 0, pos = 0; dev < numDevices; pos += domainDivision[dev], dev++)
		{
			hipSetDevice(dev);
			cout << endl << endl << "Kernal Execution on GPU : " << dev;
			cout << endl << "Position :" << pos;


			cout << endl << "Check Intermediate Result before it gets passed to kernal" << endl;

			hipMemcpy(&result[0] + pos, d_Vec_In[dev], domainDivision[dev] * sizeof(float), hipMemcpyDeviceToHost);

			for (int i = size - 1; i >= 0; i--) {


				if ((i + 1) % dim == 0) { cout << endl; }

				cout << "#pos:" << i << " " << result[i] << "    ";
			}

			jacobi_Simple <<<blocksize, threads >>>(d_A0[dev], d_A1[dev], d_A2[dev], d_A3[dev], d_A4[dev], d_Vec_In[dev], d_Vec_Out[dev], d_Rhs[dev], d_nhalos[dev], d_shalos[dev], deviceArray[dev].deviceID, numDevices);

			//TODO: Currently serial has to be done hipMemcpyAsync using CUDA Streams

			//Copy the intermediate result from Device to Host memory
			hipMemcpy(&result[0] + pos, d_Vec_Out[dev], domainDivision[dev] * sizeof(float), hipMemcpyDeviceToHost);
			//Copy the intermediate result from the Host memory to the Device memory
			hipMemcpy(d_Vec_In[dev], &result[0] + pos, domainDivision[dev] * sizeof(float), hipMemcpyHostToDevice);


			/* Store Halo positions after iteration for exchanging */
			if (numDevices > 1)
			{
				if (dev == 0) {
					hipMemcpy(&prev_nHalo[0], d_nhalos[dev], dim * sizeof(float), hipMemcpyDeviceToHost);
				}
				else if (dev == (numDevices - 1)) {
					//Exchange Happens here
					hipMemcpy(&curr_sHalo[0], d_shalos[dev], dim * sizeof(float), hipMemcpyDeviceToHost);
					hipMemcpy(d_shalos[dev], &prev_nHalo[0], dim * sizeof(float), hipMemcpyHostToDevice);
					hipMemcpy(d_nhalos[dev - 1], &curr_sHalo[0], dim * sizeof(float), hipMemcpyHostToDevice);

				}
				else {
					//Exchange Happens here
					hipMemcpy(&curr_sHalo[0], d_shalos[dev], dim * sizeof(float), hipMemcpyDeviceToHost);
					hipMemcpy(d_shalos[dev], &prev_nHalo[0], dim * sizeof(float), hipMemcpyHostToDevice);
					hipMemcpy(d_nhalos[dev - 1], &curr_sHalo[0], dim * sizeof(float), hipMemcpyHostToDevice);
					//Store current North Boundary in prev_halo for exchanging in later step
					hipMemcpy(&prev_nHalo[0], d_nhalos[dev], dim * sizeof(float), hipMemcpyDeviceToHost);
				}
			}


		}

		//TODO: Using P2P to be done later
		//exchangeHalos(numDevices,result, d_Vec_In);
		//Exchange halo logic
		//1. Prev = current nhalo
		//2. On next  iteration shalo = Prev and, Prev = nhalo.


	}

	if (auto err = hipGetLastError())
	{
		cout << "Jacobi launch failed: " << hipGetErrorString(err) << endl;
		return err;
	}

	cout << endl << "Iterations successful " << endl;

	//Copy the final result from all devices
	for (int dev = 0, pos = 0; dev < numDevices; pos += domainDivision[dev], dev++)
	{
		hipMemcpy(&result[0] + pos, d_Vec_Out[dev], domainDivision[dev] * sizeof(float), hipMemcpyDeviceToHost);
	}



	if (auto err = hipGetLastError())
	{
		cout << "Jacobi launch failed: " << hipGetErrorString(err) << endl;
		return err;
	}

	//Print result

	for (int i = size - 1; i >= 0; i--) {


		if ((i + 1) % dim == 0) { cout << endl; }

		cout << result[i] << " ";
	}
	// Freeing memory auto done by cuda deleter

	//Free memory on devices
	for (int dev = 0; dev < numDevices; dev++)
	{
		hipFree(d_A0[dev]);
		hipFree(d_A1[dev]);
		hipFree(d_A2[dev]);
		hipFree(d_A3[dev]);
		hipFree(d_A4[dev]);
		hipFree(d_Vec_In[dev]);
		hipFree(d_Vec_Out[dev]);
		hipFree(d_nhalos[dev]);
		hipFree(d_shalos[dev]);
		hipFree(d_Rhs[dev]);
	}

	cout << endl << "Device Memory free successful.";
	//Take care of dynamic mem location
	delete[] domainDivision;

	return hipSuccess;


}


int performJacobi_MultiGPU2D_Decom(unsigned int dim, unsigned int numJacobiIt, float* A0, float* A1, float* A2, float* A3, float* A4, float* rhs, float* x_in)
{


	hipError_t cudaStatus = performMultiGPUJacobi(dim, numJacobiIt, &A0[0], &A1[0], &A2[0], &A3[0], &A4[0], &rhs[0], &x_in[0]);

	if (cudaStatus != hipSuccess) {
		cout << "Computation failed: " << endl;
		return 1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		cout << "Cuda Device Reset failed: " << endl;
		return 1;
	}

	return 0;

}
