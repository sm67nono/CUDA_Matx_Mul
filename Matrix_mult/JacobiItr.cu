#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <memory>
using namespace std;
using namespace std::chrono;

#define IMUL(a,b) __mul24(a,b)


hipError_t performJacobi();


struct cuda_deleter
{
	void operator() (void * p) { hipFree(p); }
};

template<typename T>
auto make_unique_cuda_array(std::size_t size)
{
	T * p = nullptr;
	if (auto err = hipMalloc((void**)&p, size * sizeof(T)))
		throw std::bad_alloc();
	return std::unique_ptr<T[], cuda_deleter>(p);
}


//Simple Jacobi iteration
__global__ void jacobi_Simple(const float *A0, const float *A1, const float *A2, const float *A3, const float *A4, int dim, float *x, const float *rhs)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	float result = rhs[index];


	//Get the boundaries

	int leftBoundaryElem = blockDim.x * blockIdx.x;

	int rightBoundaryElem = (dim - 1) + blockDim.x * blockIdx.x;

	int topBoundaryElem = threadIdx.x + blockDim.x * dim;

	int bottomBoundaryElem = threadIdx.x;

	//Carry out computations for boundary elements
	if (index == leftBoundaryElem && index == bottomBoundaryElem) // Bottom left Corner Element
	{

		//Top
		result -= A4[index] * x[index + dim];


		//Right 

		result -= A3[index] * x[index + 1];


		result /= A2[index];

		x[index] = result;

		return;
	}

	else if (index == rightBoundaryElem && index == bottomBoundaryElem) //Bottom Right Corner Element
	{

		//Top
		result -= A4[index] * x[index + dim];

		//Left
		result -= A1[index] * x[index - 1];


		result /= A2[index];

		x[index] = result;

		return;

	}
	else if (index == leftBoundaryElem && index == topBoundaryElem) //Top left Corner Element
	{
		//Bottom
		result -= A0[index] * x[index - dim];

		//Right 

		result -= A3[index] * x[index + 1];

		result /= A2[index];

		x[index] = result;

		return;

	}

	else if (index == leftBoundaryElem && index == topBoundaryElem) //Top Right Corner Element
	{
		//Bottom
		result -= A0[index] * x[index - dim];

		//Left
		result -= A1[index] * x[index - 1];

		result /= A2[index];

		x[index] = result;

		return;

	}




	else if (index == leftBoundaryElem)
	{
		//Bottom
		result -= A0[index] * x[index - dim];

		//Top
		result -= A4[index] * x[index + dim];


		//Right 

		result -= A3[index] * x[index + 1];

		result /= A2[index];

		x[index] = result;

		return;
	}

	else if (index == bottomBoundaryElem) {


		//Top
		result -= A4[index] * x[index + dim];

		//Left
		result -= A1[index] * x[index - 1];

		//Right 

		result -= A3[index] * x[index + 1];

		result /= A2[index];

		x[index] = result;

		return;


	}

	else if (index == rightBoundaryElem) {


		//Bottom
		result -= A0[index] * x[index - dim];

		//Top
		result -= A4[index] * x[index + dim];

		//Left
		result -= A1[index] * x[index - 1];


		result /= A2[index];

		x[index] = result;

		return;


	}


	else if (index == topBoundaryElem) {


		//Bottom
		result -= A0[index] * x[index - dim];


		//Left
		result -= A1[index] * x[index - 1];

		//Right 

		result -= A3[index] * x[index + 1];

		result /= A2[index];

		x[index] = result;

		return;


	}

	//For every other element not on the boundary
	else {
		//Bottom
		result -= A0[index] * x[index - dim];

		//Top
		result -= A4[index] * x[index + dim];

		//Left
		result -= A1[index] * x[index - 1];

		//Right 

		result -= A3[index] * x[index + 1];

		result /= A2[index];

		x[index] = result;

		return;
	}

}



//Init matrix Diagonals A0, A1, A2, A3, A4
void initDiag(float *A0, float *A1, float *A2, float *A3, float *A4, float *res, float * vec, int dim)
{
	//Not accounted for Obstacles

	for (int i = 0; i < dim; ++i)
	{
		for (int j = 0; j < dim; ++j)
		{
			int idx = j + dim * i;

			//Bottom
			if (i == 0) {
				A0[idx] = 0.0f;
			}
			else {
				A0[idx] = 1.0f;
			}

			//Left 
			if (j == 0)
			{
				A1[idx] = 0.0f;
			}
			else {

				A1[idx] = 1.0f;
			}


			//Right
			if (j == dim - 1)
			{
				A3[idx] = 0.0f;
			}
			else {

				A3[idx] = 1.0f;
			}

			//Top
			if (i == dim - 1)
			{
				A4[idx] = 0.0f;
			}
			else {

				A4[idx] = 1.0f;
			}

			//Primary Diagonal 
			A2[idx] = 1.0f;

			//Result(RHS) and Vector init
			res[idx] = 1.0f;
			vec[idx] = 1.0f;


		}
	}

}




hipError_t performJacobi()
{


	//Fixed values to be changed later

	const int dim = 3;

	const int size = dim * dim;

	auto result = std::make_unique<float[]>(size);

	//Create Diagonal Vectors
	auto a0 = std::make_unique<float[]>(size);
	auto a1 = std::make_unique<float[]>(size);
	auto a2 = std::make_unique<float[]>(size);
	auto a3 = std::make_unique<float[]>(size);
	auto a4 = std::make_unique<float[]>(size);
	auto vec = std::make_unique<float[]>(size);
	auto res = std::make_unique<float[]>(size);


	initDiag(a0.get(), a1.get(), a2.get(), a3.get(), a4.get(), res.get(), vec.get(), dim);

	cout << "A0             ....";
	for (int i = 0; i < size ;i++) {
		cout << a0[i] << " ";
	}
	cout << endl;

	cout << "A1             ....";
	for (int i = 0; i < size;i++) {
		cout << a1[i] << " ";
	}
	cout << endl;
	cout << "A2             ....";
	for (int i = 0; i < size;i++) {
		cout << a2[i] << " ";
	}
	cout << endl;
	cout << "A3             ....";
	for (int i = 0; i < size;i++) {
		cout << a3[i] << " ";
	}
	cout << endl;
	cout << "A4             ....";
	for (int i = 0; i < size;i++) {
		cout << a4[i] << " ";
	}
	cout << endl;

	cout << "RHS             ....";
	for (int i = 0; i < size;i++) {
		cout << res[i] << " ";
	}
	cout << endl;

	cout << "Vec             ....";
	for (int i = 0; i < size;i++) {
		cout << vec[i] << " ";
	}
	cout << endl;



	//For use on Device 
	auto d_A0 = make_unique_cuda_array<float>(size);
	auto d_A1 = make_unique_cuda_array<float>(size);
	auto d_A2 = make_unique_cuda_array<float>(size);
	auto d_A3 = make_unique_cuda_array<float>(size);
	auto d_A4 = make_unique_cuda_array<float>(size);
	auto d_Vec = make_unique_cuda_array<float>(size);
	auto d_Res = make_unique_cuda_array<float>(size);

	if (auto err = hipGetLastError())
	{
		fprintf(stderr, "Jacobi launch failed: %s\n", hipGetErrorString(err));
		return err;
	}

	//cudamalloc the Diagonals
	hipMalloc((void**)&d_A0, size * sizeof(float));
	hipMalloc((void**)&d_A1, size * sizeof(float));
	hipMalloc((void**)&d_A2, size * sizeof(float));
	hipMalloc((void**)&d_A3, size * sizeof(float));
	hipMalloc((void**)&d_A4, size * sizeof(float));

	//cudamalloc the Input Vector and Result vector
	hipMalloc((void**)&d_Vec, size * sizeof(float));
	hipMalloc((void**)&d_Res, size * sizeof(float));

	if (auto err = hipGetLastError())
	{
		fprintf(stderr, "Jacobi launch failed: %s\n", hipGetErrorString(err));
		return err;
	}


	hipMemcpy(d_A0.get(), a0.get(), size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_A1.get(), a1.get(), size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_A2.get(), a2.get(), size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_A3.get(), a3.get(), size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_A4.get(), a4.get(), size * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(d_Vec.get(), vec.get(), size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Res.get(), res.get(), size * sizeof(float), hipMemcpyHostToDevice);

	if (auto err = hipGetLastError())
	{
		fprintf(stderr, "Jacobi launch failed: %s\n", hipGetErrorString(err));
		return err;
	}

	//multMatrix(d_A0, d_A1, d_A2, d_A3, d_A4, myDim, d_vec, d_res);

	//Perform one Jacobi Step
	int blocksize = 5;
	int threads = 5;


	jacobi_Simple <<<blocksize, threads>>>(d_A0.get(), d_A1.get(), d_A2.get(), d_A3.get(), d_A4.get(), dim, d_Vec.get(), d_Res.get());

	if (auto err = hipGetLastError())
	{
		fprintf(stderr, "Jacobi launch failed: %s\n", hipGetErrorString(err));
		return err;
	}

	hipMemcpy(result.get(), d_Vec.get(), size * sizeof(float), hipMemcpyDeviceToHost);

	if (auto err = hipGetLastError())
	{
		fprintf(stderr, "Jacobi launch failed: %s\n", hipGetErrorString(err));
		return err;
	}


	cout << "One iteration successful";

	//Print result
	for (int i = 0; i < size; i++) {


		if (i % dim == 0) { cout << endl; }

		cout << vec[i] << " ";
	}

	cout << endl << endl;
	for (int i = 0; i < size; i++) {


		if (i % dim == 0) { cout << endl; }

		cout << result[i] << " ";
	}
	// Freeing memory auto done by cuda deleter

	/*cudaFree(d_A0.get());
	cudaFree(d_A1.get());
	cudaFree(d_A2.get());
	cudaFree(d_A3.get());
	cudaFree(d_A4.get());
	cudaFree(d_Vec.get());
	cudaFree(d_Res.get());*/

	return hipSuccess;


}


int main()
{
	

	hipError_t cudaStatus = performJacobi();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Computation failed! \n");
		return 1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed! \n");
		return 1;
	}

	return 0;

}
