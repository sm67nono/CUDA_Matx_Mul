#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<iostream>
#include<chrono>
using namespace std;
using namespace std::chrono;

hipError_t performNormalMatrixMultiplication();

__global__ void multiply(float *dev_a, float *dev_x, float *dev_b)
{
	int i = blockIdx.x;
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	dev_b[i] = dev_a[index] * dev_x[i]; //Row multiplication of matrix A with vector x
}

//Later init can be moved to GPU
void initArrays(float *a, float *x, float *b) 
{
	int index = 0;
	for (int i = 0; i < 32; i++) {
		x[i] = i*0.56;
		b[i] = 0.0;
		for (int j = 0; j < 32; j++) {
			a[index] = i * j * 0.045 * (index/89); //Generating a random number and storing in a[index]
			index++;
		}
		
	}
}

hipError_t performNormalMatrixMultiplication()
{
	int size = 32;
	//Create Matrix Vectors
	float c[32];//To copy final result from device to host

	float *a = new float[1024]; //Total elements in one matrix 32 x 32
	float *x = new float[32]; //Vector to be multiplied
	float *b = new float[32]; //Resultant vector

	//For use on Device 
	float *dev_a, *dev_x, *dev_b;

	initArrays(a,x,b);
	cout << sizeof(*a);
	cout << sizeof(*b);
	cout << sizeof(*x);

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}



	// Allocate GPU buffers for three vectors (two input, one output)    
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, 1024 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_x, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, 1024 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_x, x, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	//To refer each element of the matrix we get 32 blocks with 32 threads
	int blocksize = 32;
	int gridsize = 32;
	printf("The gridsize is %d", gridsize);
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	multiply <<<gridsize, blocksize >>>(a,x,b);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "multiply launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	//After everything is syncronized

	high_resolution_clock::time_point t2 = high_resolution_clock::now();

	auto duration = duration_cast<microseconds>(t2 - t1).count();

	cout << "Duration to execute the parallel portion is " << duration << endl;


	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, &dev_b, 32 * sizeof(float), hipMemcpyDeviceToHost);

	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_b);
	return cudaStatus;
}






int main()
{
	hipError_t cudaStatus = performNormalMatrixMultiplication(); 



	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Matrix Multiply failed!");
		return 1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;

}

	
